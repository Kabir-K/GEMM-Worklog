#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define M 4096  
#define K 4096 
#define N 4096
#define BLOCK_SIZE 32



__global__ void matmul_gpu(float *A, float *B, float *C, int m, int k, int n){

  const int cRow = blockIdx.x * BLOCK_SIZE + (threadIdx.x / BLOCK_SIZE);
  const int cCol = blockIdx.y * BLOCK_SIZE + (threadIdx.x % BLOCK_SIZE);
  
  if (cRow < m && cCol < n) {
    float tmp = 0.0;
    for (int i = 0; i < k; ++i) {
      tmp += A[cRow * k + i] * B[i * n + cCol];
    }
    C[cRow * n + cCol] = tmp;
  }
}

void init_matrix(float *mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        mat[i] = (float)rand() / RAND_MAX;
    }
}


double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main() {
    float *h_A, *h_B, *h_C_cpu, *h_C_gpu;
    float *d_A, *d_B, *d_C;
    int size_A = M * K * sizeof(float);
    int size_B = K * N * sizeof(float);
    int size_C = M * N * sizeof(float);

    h_A = (float*)malloc(size_A);
    h_B = (float*)malloc(size_B);
    h_C_cpu = (float*)malloc(size_C);
    h_C_gpu = (float*)malloc(size_C);

    srand(time(NULL));
    init_matrix(h_A, M, K);
    init_matrix(h_B, K, N);

 
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);


    for (int i = 0; i < 3; i++) {
        matmul_gpu<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, K, N);
        hipDeviceSynchronize();
    }

    double total = 0;
    int runs = 20;
    for (int i = 0; i < runs; i++) {
        double start_time = get_time();
        matmul_gpu<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, K, N);
        hipDeviceSynchronize();
        double end_time = get_time();
        total += end_time - start_time;
    }

    double avg_s = total / runs;
    double avg_ms = avg_s * 1000;

    double total_flops = 2.0 * (double)M * (double)N * (double)K;
    double gflops = total_flops / (avg_s * 1.0e9);
    double tflops = gflops/1000;
    
    printf(
        "M=%d N=%d K=%d | runs=%d\n"
        "Avg time = %.3f ms (%.6f s) | GFLOPS = %.2f | TFLOPs = %.2f\n",
        M, N, K, runs,
        avg_ms, avg_s, gflops, tflops
    );

    free(h_A);
    free(h_B);
    free(h_C_cpu);
    free(h_C_gpu);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

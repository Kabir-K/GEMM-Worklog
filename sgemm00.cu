#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#define M 4096
#define K 4096
#define N 4096



#define CHECK_CUBLAS(call) { \
    hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS error in %s:%d: %d\n", __FILE__, __LINE__, status); \
        exit(EXIT_FAILURE); \
    } \
}


void initialize(float *a, int size){
    for(int i=0; i<size; i++){
        a[i] = i%1000;
    }
}

int main(){
    float *h_a = (float*) malloc(M * K * sizeof(float));
    float *h_b = (float*) malloc(K * N * sizeof(float));

    initialize(h_a,M*K);
    initialize(h_b,N*K);

    float *d_a, *d_b, *d_c;

    hipMalloc(&d_a, M * K * sizeof(float));
    hipMalloc(&d_b, N * K * sizeof(float));
    hipMalloc(&d_c, M * N * sizeof(float));

    hipMemcpy(d_a, h_a, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, K * N * sizeof(float), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    float alpha = 1.0f, beta = 0.0f;

    for(int i=0; i<10; i++){
        CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_b, N, d_a, K, &beta, d_c, N));
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    double total = 0;
    int runs = 20;
    for(int i=0; i<runs; i++){
        hipEventRecord(start);
        CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_b, N, d_a, K, &beta, d_c, N));
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms;
        hipEventElapsedTime(&ms,start, stop);    
        total += ms;    
    }
    
    double avg_ms = total / runs;
    double avg_s = avg_ms / 1000;

    double total_flops = 2.0 * (double)M * (double)N * (double)K;
    double gflops = total_flops / (avg_s * 1.0e9);
    double tflops = gflops/1000;
    
    printf(
        "M=%d N=%d K=%d | runs=%d\n"
        "Avg time = %.3f ms (%.6f s) | GFLOPS = %.2f | TFLOPs = %.2f\n",
        M, N, K, runs,
        avg_ms, avg_s, gflops, tflops
    );
    CHECK_CUBLAS(hipblasDestroy(handle));

}

